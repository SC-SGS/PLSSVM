#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/cg_explicit/kernel_matrix_assembly.cuh"

#include "plssvm/constants.hpp"  // plssvm::real_type, plssvm::kernel_index_type

namespace plssvm::cuda {

__global__ void device_kernel_assembly_linear(const real_type *q, real_type *ret, const real_type *data_d, const real_type QA_cost, const real_type cost, const kernel_index_type dept, const kernel_index_type num_features) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < dept && j < dept) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; ++dim) {
            temp += data_d[i * num_features + dim] * data_d[j * num_features + dim];
        }
        temp = temp + QA_cost - q[i] - q[j];
        if (i == j) {
            temp += cost;
        }

        ret[i * dept + j] = temp;
    }
}

__global__ void device_kernel_assembly_polynomial(const real_type *q, real_type *ret, const real_type *data_d, const real_type QA_cost, const real_type cost, const kernel_index_type num_rows, const kernel_index_type num_features, const int degree, const real_type gamma, const real_type coef0) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < num_rows && j < num_rows) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; ++dim) {
            temp += data_d[i * num_features + dim] * data_d[j * num_features + dim];
        }
        temp = pow(gamma * temp + coef0, degree) + QA_cost - q[i] - q[j];
        if (i == j) {
            temp += cost;
        }

        ret[i * num_rows + j] = temp;
    }
}

__global__ void device_kernel_assembly_rbf(const real_type *q, real_type *ret, const real_type *data_d, const real_type QA_cost, const real_type cost, const kernel_index_type num_rows, const kernel_index_type num_features, const real_type gamma) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < num_rows && j < num_rows) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; ++dim) {
            const real_type d = data_d[i * num_features + dim] - data_d[j * num_features + dim];
            temp += d * d;
        }
        temp = exp(-gamma * temp) + QA_cost - q[i] - q[j];
        if (i == j) {
            temp += cost;
        }

        ret[i * num_rows + j] = temp;
    }
}

}  // namespace plssvm::cuda