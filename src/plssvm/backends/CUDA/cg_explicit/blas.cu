#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/cg_explicit/blas.cuh"

#include "plssvm/constants.hpp"  // plssvm::real_type, plssvm::THREAD_BLOCK_SIZE, plssvm::FEATURE_BLOCK_SIZE

namespace plssvm::cuda {

__global__ void device_kernel_gemm(const unsigned long long m, const unsigned long long n, const unsigned long long k, const real_type alpha, const real_type *A, const real_type *B, const real_type beta, real_type *C) {
    // compute: C = alpha * A * B + beta * C with A in m x k, B in n x k, and C in n x m, alpha, beta as scalar
    const unsigned long long i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;  // # rhs
    const unsigned long long i_linear = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE + threadIdx.x;
    const unsigned long long j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;  // # rows
    const unsigned long long j_cached_idx_linear = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE + threadIdx.x;

    __shared__ real_type A_cache[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];
    __shared__ real_type B_cache[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];

    real_type temp[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };

    for (unsigned long long dim = 0; dim < k; dim += FEATURE_BLOCK_SIZE) {
        // zero out shared memory
        for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
            A_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            A_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            B_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            B_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
        }

        // load data into shared memory
        for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
            const unsigned long long global_i = i_linear + internal * THREAD_BLOCK_SIZE;
            const unsigned long long global_j = j_cached_idx_linear + internal * THREAD_BLOCK_SIZE;

            if (global_j < k) {
                if (dim + threadIdx.y < k) {
                    A_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = A[(dim + threadIdx.y) * k + global_j];
                }
                if (dim + threadIdx.y + THREAD_BLOCK_SIZE < k) {
                    A_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = A[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * k + global_j];
                }
            }

            if (global_i < n) {
                if (dim + threadIdx.y < k) {
                    B_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = B[(dim + threadIdx.y) * n + global_i];
                }
                if (dim + threadIdx.y + THREAD_BLOCK_SIZE < k) {
                    B_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = B[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * n + global_i];
                }
            }
        }
        __syncthreads();

        // calculation
        for (unsigned block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
            for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
                for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                    temp[internal_i][internal_j] += A_cache[block_dim][threadIdx.y * INTERNAL_BLOCK_SIZE + internal_j] * B_cache[block_dim][threadIdx.x * INTERNAL_BLOCK_SIZE + internal_i];
                }
            }
        }
        __syncthreads();
    }

    for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
        for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
            const unsigned long long global_i = i + internal_i;
            const unsigned long long global_j = j + internal_j;

            if (global_i < n && global_j < m) {
                C[global_j * n + global_i] = alpha * temp[internal_i][internal_j] + beta * C[global_j * n + global_i];
            }
        }
    }
}

__global__ void device_kernel_symm(const unsigned long long m, const unsigned long long n, const unsigned long long k, const real_type alpha, const real_type *A, const real_type *B, const real_type beta, real_type *C) {
    // compute: C = alpha * A * B + beta * C with A in m x k, B in n x k, and C in n x m, alpha, beta as scalar
    const unsigned long long i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;  // # rhs
    const unsigned long long i_linear = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE + threadIdx.x;
    const unsigned long long j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;  // # rows
    const unsigned long long j_cached_idx_linear = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE + threadIdx.x;

    __shared__ real_type A_cache[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];
    __shared__ real_type B_cache[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];

    real_type temp[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };

    for (unsigned long long dim = 0; dim < k; dim += FEATURE_BLOCK_SIZE) {
        // zero out shared memory
        for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
            A_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            A_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            B_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            B_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
        }

        // load data into shared memory
        for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
            const unsigned long long global_i = i_linear + internal * THREAD_BLOCK_SIZE;
            const unsigned long long global_j = j_cached_idx_linear + internal * THREAD_BLOCK_SIZE;

            if (dim + threadIdx.y < k) {
                if (dim + threadIdx.y < global_j) {
                    if (global_j < k) {
                        A_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = A[(dim + threadIdx.y) * k + global_j - (dim + threadIdx.y) * (dim + threadIdx.y + 1) / 2];
                    }
                } else {
                    A_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = A[global_j * k + dim + threadIdx.y - global_j * (global_j + 1) / 2];
                }
            }

            if (dim + threadIdx.y + THREAD_BLOCK_SIZE < k) {
                if (dim + threadIdx.y + THREAD_BLOCK_SIZE < global_j) {
                    if (global_j < k) {
                        A_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = A[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * k + global_j - (dim + threadIdx.y + THREAD_BLOCK_SIZE) * (dim + threadIdx.y + THREAD_BLOCK_SIZE + 1) / 2];
                    }
                } else {
                    A_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = A[global_j * k + dim + threadIdx.y + THREAD_BLOCK_SIZE - global_j * (global_j + 1) / 2];
                }
            }

            if (global_i < n) {
                if (dim + threadIdx.y < k) {
                    B_cache[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = B[(dim + threadIdx.y) * n + global_i];
                }
                if (dim + threadIdx.y + THREAD_BLOCK_SIZE < k) {
                    B_cache[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = B[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * n + global_i];
                }
            }
        }
        __syncthreads();

        // calculation
        for (unsigned block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
            for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
                for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                    temp[internal_i][internal_j] += A_cache[block_dim][threadIdx.y * INTERNAL_BLOCK_SIZE + internal_j] * B_cache[block_dim][threadIdx.x * INTERNAL_BLOCK_SIZE + internal_i];
                }
            }
        }
        __syncthreads();
    }

    for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
        for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
            const unsigned long long global_i = i + internal_i;
            const unsigned long long global_j = j + internal_j;

            if (global_i < n && global_j < m) {
                C[global_j * n + global_i] = alpha * temp[internal_i][internal_j] + beta * C[global_j * n + global_i];
            }
        }
    }
}

}  // namespace plssvm::cuda