/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/csvm.hpp"

#include "plssvm/backends/CUDA/detail/device_ptr.cuh"  // plssvm::cuda::detail::device_ptr
#include "plssvm/backends/CUDA/detail/utility.cuh"     // plssvm::cuda::detail::{device_synchronize, get_device_count, set_device, peek_at_last_error}
#include "plssvm/backends/CUDA/exceptions.hpp"         // plssvm::cuda::backend_exception
#include "plssvm/backends/CUDA/predict_kernel.cuh"     // plssvm::cuda::detail::{device_kernel_w_linear, device_kernel_predict_polynomial, device_kernel_predict_rbf}
#include "plssvm/backends/CUDA/q_kernel.cuh"           // plssvm::cuda::detail::{device_kernel_q_linear, device_kernel_q_polynomial, device_kernel_q_rbf}
#include "plssvm/backends/CUDA/svm_kernel.cuh"         // plssvm::cuda::detail::{device_kernel_linear, device_kernel_polynomial, device_kernel_rbf}
#include "plssvm/detail/assert.hpp"                    // PLSSVM_ASSERT
#include "plssvm/detail/execution_range.hpp"           // plssvm::detail::execution_range
#include "plssvm/exceptions/exceptions.hpp"            // plssvm::exception
#include "plssvm/kernel_function_types.hpp"            // plssvm::kernel_function_type
#include "plssvm/parameter.hpp"                        // plssvm::parameter, plssvm::detail::parameter
#include "plssvm/target_platforms.hpp"                 // plssvm::target_platform

#include "hip/hip_runtime.h"              // cuda runtime functions
#include "hip/hip_runtime_api.h"  // cuda runtime functions

#include "fmt/core.h"     // fmt::format
#include "fmt/ostream.h"  // can use fmt using operator<< overloads

#include <cstddef>    // std::size_t
#include <exception>  // std::terminate
#include <iostream>   // std::cout, std::endl
#include <numeric>    // std::iota
#include <utility>    // std::pair, std::make_pair

namespace plssvm::cuda {

csvm::csvm(parameter params) :
    csvm{ plssvm::target_platform::automatic, params } {}

csvm::csvm(target_platform target, parameter params) :
    base_type{ params } {
    this->init(target);
}

void csvm::init(const target_platform target) {
    // check if supported target platform has been selected
    if (target != target_platform::automatic && target != target_platform::gpu_nvidia) {
        throw backend_exception{ fmt::format("Invalid target platform '{}' for the CUDA backend!", target) };
    } else {
#if !defined(PLSSVM_HAS_NVIDIA_TARGET)
        throw backend_exception{ "Requested target platform 'gpu_nvidia' that hasn't been enabled using PLSSVM_TARGET_PLATFORMS!" };
#endif
    }

    if (plssvm::verbose) {
        std::cout << fmt::format("\nUsing CUDA as backend.") << std::endl;
    }

    // get all available devices wrt the requested target platform
    devices_.resize(detail::get_device_count());
    std::iota(devices_.begin(), devices_.end(), 0);

    // throw exception if no CUDA devices could be found
    if (devices_.empty()) {
        throw backend_exception{ "CUDA backend selected but no CUDA capable devices were found!" };
    }

    if (plssvm::verbose) {
        // print found CUDA devices
        std::cout << fmt::format("Found {} CUDA device(s):\n", devices_.size());
        for (const queue_type &device : devices_) {
            hipDeviceProp_t prop{};
            hipGetDeviceProperties(&prop, device);
            std::cout << fmt::format("  [{}, {}, {}.{}]\n", device, prop.name, prop.major, prop.minor) << std::endl;
        }
        std::cout << std::endl;
    }
}

csvm::~csvm() {
    try {
        // be sure that all operations on the CUDA devices have finished before destruction
        for (const queue_type &device : devices_) {
            detail::device_synchronize(device);
        }
    } catch (const plssvm::exception &e) {
        std::cout << e.what_with_loc() << std::endl;
        std::terminate();
    }
}

void csvm::device_synchronize(const queue_type &queue) const {
    detail::device_synchronize(queue);
}

std::pair<dim3, dim3> execution_range_to_native(const ::plssvm::detail::execution_range &range) {
    dim3 grid(range.grid[0], range.grid[1], range.grid[2]);
    dim3 block(range.block[0], range.block[1], range.block[2]);
    return std::make_pair(grid, block);
}

template <typename real_type>
void csvm::run_q_kernel_impl(const std::size_t device, const ::plssvm::detail::execution_range &range, const ::plssvm::detail::parameter<real_type> &params, device_ptr_type<real_type> &q_d, const device_ptr_type<real_type> &data_d, const device_ptr_type<real_type> &data_last_d, const std::size_t num_data_points_padded, const std::size_t num_features) const {
    const auto [grid, block] = execution_range_to_native(range);

    detail::set_device(static_cast<queue_type>(device));
    switch (params.kernel_type) {
        case kernel_function_type::linear:
            cuda::device_kernel_q_linear<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features));
            break;
        case kernel_function_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_polynomial<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_function_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_rbf<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
}

template void csvm::run_q_kernel_impl(std::size_t, const ::plssvm::detail::execution_range &, const ::plssvm::detail::parameter<float> &, device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, std::size_t, std::size_t) const;
template void csvm::run_q_kernel_impl(std::size_t, const ::plssvm::detail::execution_range &, const ::plssvm::detail::parameter<double> &, device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, std::size_t, std::size_t) const;

template <typename real_type>
void csvm::run_svm_kernel_impl(const std::size_t device, const ::plssvm::detail::execution_range &range, const ::plssvm::detail::parameter<real_type> &params, const device_ptr_type<real_type> &q_d, device_ptr_type<real_type> &r_d, const device_ptr_type<real_type> &x_d, const device_ptr_type<real_type> &data_d, const real_type QA_cost, const real_type add, const std::size_t num_data_points_padded, const std::size_t num_features) const {
    const auto [grid, block] = execution_range_to_native(range);

    detail::set_device(static_cast<queue_type>(device));
    switch (params.kernel_type) {
        case kernel_function_type::linear:
            cuda::device_kernel_linear<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d.get(), QA_cost, 1 / params.cost, static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), add, static_cast<kernel_index_type>(device));
            break;
        case kernel_function_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_polynomial<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d.get(), QA_cost, 1 / params.cost, static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), add, params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_function_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_rbf<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d.get(), QA_cost, 1 / params.cost, static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), add, params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
}

template void csvm::run_svm_kernel_impl(std::size_t, const ::plssvm::detail::execution_range &, const ::plssvm::detail::parameter<float> &, const device_ptr_type<float> &, device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, float, float, std::size_t, std::size_t) const;
template void csvm::run_svm_kernel_impl(std::size_t, const ::plssvm::detail::execution_range &, const ::plssvm::detail::parameter<double> &, const device_ptr_type<double> &, device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, double, double, std::size_t, std::size_t) const;

template <typename real_type>
void csvm::run_w_kernel_impl(const std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type<real_type> &w_d, const device_ptr_type<real_type> &alpha_d, const device_ptr_type<real_type> &data_d, const device_ptr_type<real_type> &data_last_d, const std::size_t num_data_points, const std::size_t num_features) const {
    const auto [grid, block] = execution_range_to_native(range);

    detail::set_device(static_cast<queue_type>(device));
    cuda::device_kernel_w_linear<<<grid, block>>>(w_d.get(), data_d.get(), data_last_d.get(), alpha_d.get(), static_cast<kernel_index_type>(num_data_points), static_cast<kernel_index_type>(num_features));
    detail::peek_at_last_error();
}

template void csvm::run_w_kernel_impl(std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, std::size_t, std::size_t) const;
template void csvm::run_w_kernel_impl(std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, std::size_t, std::size_t) const;

template <typename real_type>
void csvm::run_predict_kernel_impl(const ::plssvm::detail::execution_range &range, const ::plssvm::detail::parameter<real_type> &params, device_ptr_type<real_type> &out_d, const device_ptr_type<real_type> &alpha_d, const device_ptr_type<real_type> &point_d, const device_ptr_type<real_type> &data_d, const device_ptr_type<real_type> &data_last_d, const std::size_t num_support_vectors, const std::size_t num_predict_points, const std::size_t num_features) const {
    const auto [grid, block] = execution_range_to_native(range);

    detail::set_device(0);
    switch (params.kernel_type) {
        case kernel_function_type::linear:
            break;
        case kernel_function_type::polynomial:
            cuda::device_kernel_predict_polynomial<<<grid, block>>>(out_d.get(), data_d.get(), data_last_d.get(), alpha_d.get(), static_cast<kernel_index_type>(num_support_vectors), point_d.get(), static_cast<kernel_index_type>(num_predict_points), static_cast<kernel_index_type>(num_features), params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_function_type::rbf:
            cuda::device_kernel_predict_rbf<<<grid, block>>>(out_d.get(), data_d.get(), data_last_d.get(), alpha_d.get(), static_cast<kernel_index_type>(num_support_vectors), point_d.get(), static_cast<kernel_index_type>(num_predict_points), static_cast<kernel_index_type>(num_features), params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
}

template void csvm::run_predict_kernel_impl(const ::plssvm::detail::execution_range &, const ::plssvm::detail::parameter<float> &, device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, std::size_t, std::size_t, std::size_t) const;
template void csvm::run_predict_kernel_impl(const ::plssvm::detail::execution_range &, const ::plssvm::detail::parameter<double> &, device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, std::size_t, std::size_t, std::size_t) const;

}  // namespace plssvm::cuda
