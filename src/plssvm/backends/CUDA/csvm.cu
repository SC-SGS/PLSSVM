#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/csvm.hpp"

#include "plssvm/backends/CUDA/detail/device_ptr.cuh"  // plssvm::cuda::detail::device_ptr
#include "plssvm/backends/CUDA/detail/utility.cuh"     // plssvm::cuda::detail::{device_synchronize, get_device_count, set_device, peek_at_last_error}
#include "plssvm/backends/CUDA/exceptions.hpp"         // plssvm::cuda::backend_exception
#include "plssvm/backends/CUDA/predict_kernel.cuh"     // plssvm::cuda::detail::{device_kernel_w_linear, device_kernel_predict_polynomial, device_kernel_predict_rbf}
#include "plssvm/backends/CUDA/q_kernel.cuh"           // plssvm::cuda::detail::{device_kernel_q_linear, device_kernel_q_polynomial, device_kernel_q_rbf}
#include "plssvm/backends/CUDA/svm_kernel.cuh"         // plssvm::cuda::detail::{device_kernel_linear, device_kernel_polynomial, device_kernel_rbf}
#include "plssvm/detail/assert.hpp"                    // PLSSVM_ASSERT
#include "plssvm/detail/execution_range.hpp"           // plssvm::detail::execution_range
#include "plssvm/detail/logger.hpp"                    // plssvm::detail::log, plssvm::verbosity_level
#include "plssvm/detail/performance_tracker.hpp"       // plssvm::detail::tracking_entry
#include "plssvm/exceptions/exceptions.hpp"            // plssvm::exception
#include "plssvm/kernel_function_types.hpp"            // plssvm::kernel_function_type
#include "plssvm/parameter.hpp"                        // plssvm::parameter, plssvm::detail::parameter
#include "plssvm/target_platforms.hpp"                 // plssvm::target_platform

#include "plssvm/backends/CUDA/blas.cuh"
#include "plssvm/backends/CUDA/kernel_matrix_assembly.cuh"

#include "hip/hip_runtime.h"                                      // cuda runtime functions
#include "hip/hip_runtime_api.h"                          // cuda runtime functions

#include "fmt/core.h"                                  // fmt::format
#include "fmt/ostream.h"                               // can use fmt using operator<< overloads

#include <cstddef>                                     // std::size_t
#include <exception>                                   // std::terminate
#include <iostream>                                    // std::cout, std::endl
#include <numeric>                                     // std::iota
#include <utility>                                     // std::pair, std::make_pair

namespace plssvm::cuda {

csvm::csvm(parameter params) :
    csvm{ plssvm::target_platform::automatic, params } {}

csvm::csvm(target_platform target, parameter params) :
    base_type{ params } {
    this->init(target);
}

void csvm::init(const target_platform target) {
    // check if supported target platform has been selected
    if (target != target_platform::automatic && target != target_platform::gpu_nvidia) {
        throw backend_exception{ fmt::format("Invalid target platform '{}' for the CUDA backend!", target) };
    } else {
#if !defined(PLSSVM_HAS_NVIDIA_TARGET)
        throw backend_exception{ "Requested target platform 'gpu_nvidia' that hasn't been enabled using PLSSVM_TARGET_PLATFORMS!" };
#endif
    }

    plssvm::detail::log(verbosity_level::full,
                        "\nUsing CUDA as backend.\n");
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "backend", plssvm::backend_type::cuda }));
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "target_platform", plssvm::target_platform::gpu_nvidia }));

    // update the target platform
    target_ = plssvm::target_platform::gpu_nvidia;

    // get all available devices wrt the requested target platform
    devices_.resize(detail::get_device_count());
    std::iota(devices_.begin(), devices_.end(), 0);

    // throw exception if no CUDA devices could be found
    if (devices_.empty()) {
        throw backend_exception{ "CUDA backend selected but no CUDA capable devices were found!" };
    }

    // print found CUDA devices
    plssvm::detail::log(verbosity_level::full,
                        "Found {} CUDA device(s):\n", plssvm::detail::tracking_entry{ "backend", "num_devices", devices_.size() });
    for (const queue_type &device : devices_) {
        hipDeviceProp_t prop{};
        hipGetDeviceProperties(&prop, device);
        plssvm::detail::log(verbosity_level::full,
                            "  [{}, {}, {}.{}]\n\n", device, prop.name, prop.major, prop.minor);
    }
    plssvm::detail::log(verbosity_level::full | verbosity_level::timing,
                        "\n");
}

csvm::~csvm() {
    try {
        // be sure that all operations on the CUDA devices have finished before destruction
        for (const queue_type &device : devices_) {
            detail::device_synchronize(device);
        }
    } catch (const plssvm::exception &e) {
        std::cout << e.what_with_loc() << std::endl;
        std::terminate();
    }
}

void csvm::device_synchronize(const queue_type &queue) const {
    detail::device_synchronize(queue);
}

std::pair<dim3, dim3> execution_range_to_native(const ::plssvm::detail::execution_range &range) {
    const dim3 grid(range.grid[0], range.grid[1], range.grid[2]);
    const dim3 block(range.block[0], range.block[1], range.block[2]);
    return std::make_pair(grid, block);
}


template <typename real_type>
auto csvm::run_w_kernel_impl(const device_ptr_type<real_type> &alpha_d, const device_ptr_type<real_type> &sv_d, std::size_t num_classes, std::size_t num_sv, std::size_t num_features) const -> device_ptr_type<real_type> {
    device_ptr_type<real_type> w_d{ num_classes * num_features };

    const dim3 block(256, 4);
    const dim3 grid(static_cast<int>(std::ceil(num_features / static_cast<double>(block.x))),
                    static_cast<int>(std::ceil(num_classes / static_cast<double>(block.y))));

    detail::set_device(0);
    cuda::device_kernel_w_linear<<<grid, block>>>(w_d.get(), alpha_d.get(), sv_d.get(), num_classes, num_sv, num_features);
    detail::peek_at_last_error();

    return w_d;
}

template auto csvm::run_w_kernel_impl(const device_ptr_type<float> &, const device_ptr_type<float> &, std::size_t, std::size_t, std::size_t) const -> device_ptr_type<float>;
template auto csvm::run_w_kernel_impl(const device_ptr_type<double> &, const device_ptr_type<double> &, std::size_t, std::size_t, std::size_t) const -> device_ptr_type<double>;


template <typename real_type>
auto csvm::run_predict_kernel_impl(const ::plssvm::detail::parameter<real_type> &params, const device_ptr_type<real_type> &w_d, const device_ptr_type<real_type> &alpha_d, const device_ptr_type<real_type> &rho_d, const device_ptr_type<real_type> &sv_d, const device_ptr_type<real_type> &predict_points_d, std::size_t num_classes, std::size_t num_sv, std::size_t num_predict_points, std::size_t num_features) const -> device_ptr_type<real_type> {
    device_ptr_type<real_type> out_d{ num_predict_points * num_classes };

    detail::set_device(0);
    if (params.kernel_type == kernel_function_type::linear) {
        const dim3 block(256, 4);
        const dim3 grid(static_cast<int>(std::ceil(num_predict_points / static_cast<double>(block.x))),
                        static_cast<int>(std::ceil(num_classes / static_cast<double>(block.y))));

        cuda::device_kernel_predict_linear<<<grid, block>>>(out_d.get(), w_d.get(), rho_d.get(), predict_points_d.get(), num_classes, num_predict_points, num_features);
    } else {
        const dim3 block(16, 16, 4);
        const dim3 grid(static_cast<int>(std::ceil(num_sv / static_cast<double>(block.x))),
                        static_cast<int>(std::ceil(num_predict_points / static_cast<double>(block.y))),
                        static_cast<int>(std::ceil(num_classes / static_cast<double>(block.z))));

        switch (params.kernel_type) {
            case kernel_function_type::linear:
                // already handled
                break;
            case kernel_function_type::polynomial:
                cuda::device_kernel_predict_polynomial<<<grid, block>>>(out_d.get(), alpha_d.get(), rho_d.get(), sv_d.get(), predict_points_d.get(), static_cast<kernel_index_type>(num_classes), static_cast<kernel_index_type>(num_sv), static_cast<kernel_index_type>(num_predict_points), static_cast<kernel_index_type>(num_features), params.degree.value(), params.gamma.value(), params.coef0.value());
                break;
            case kernel_function_type::rbf:
                cuda::device_kernel_predict_rbf<<<grid, block>>>(out_d.get(), alpha_d.get(), rho_d.get(), sv_d.get(), predict_points_d.get(), static_cast<kernel_index_type>(num_classes), static_cast<kernel_index_type>(num_sv), static_cast<kernel_index_type>(num_predict_points), static_cast<kernel_index_type>(num_features), params.gamma.value());
                break;
        }
    }

    detail::peek_at_last_error();

    return out_d;
}

template auto csvm::run_predict_kernel_impl(const ::plssvm::detail::parameter<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, const device_ptr_type<float> &, std::size_t, std::size_t, std::size_t, std::size_t) const -> device_ptr_type<float>;
template auto csvm::run_predict_kernel_impl(const ::plssvm::detail::parameter<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, const device_ptr_type<double> &, std::size_t, std::size_t, std::size_t, std::size_t) const -> device_ptr_type<double>;



template <typename real_type>
void csvm::setup_data_on_devices_impl(const aos_matrix<real_type> &A) {
    const std::size_t num_rows_reduced = A.num_rows() - 1;
    const std::size_t num_features = A.num_cols();

    // initialize data_last on device
    device_ptr_type<real_type> data_last_d{ num_features };
    data_last_d.copy_to_device(A.data() +  (A.num_rows() - 1) * num_features, 0, num_features);  // pos, count
    data_last_d_ = std::move(data_last_d);

    device_ptr_type<real_type> data_d{ num_rows_reduced * num_features };
    data_d.copy_to_device(A.data(), 0, data_d.size());
    data_d_ = std::move(data_d);
}

template void csvm::setup_data_on_devices_impl(const aos_matrix<float> &);
template void csvm::setup_data_on_devices_impl(const aos_matrix<double> &);

template <typename real_type>
std::vector<real_type> csvm::generate_q2_impl(const ::plssvm::detail::parameter<real_type> &params, const std::size_t num_data_points_reduced, const std::size_t num_features) {
    const std::size_t device = 0;// TODO: implement

    const device_ptr_type<real_type> &data_d = std::get<device_ptr_type<real_type>>(data_d_);
    const device_ptr_type<real_type> &data_last_d = std::get<device_ptr_type<real_type>>(data_last_d_);

    const dim3 block(1024);
    const dim3 grid(static_cast<int>(std::ceil(num_data_points_reduced / static_cast<double>(block.x))));

    detail::set_device(static_cast<queue_type>(device));
    device_ptr_type<real_type> q_d{ num_data_points_reduced };
    switch (params.kernel_type) {
        case kernel_function_type::linear:
            cuda::device_kernel_q_linear<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_reduced), static_cast<kernel_index_type>(num_features));
            break;
        case kernel_function_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_polynomial<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_reduced), static_cast<kernel_index_type>(num_features), params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_function_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_rbf<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_reduced), static_cast<kernel_index_type>(num_features), params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
    detail::device_synchronize(device);

    // return host array
    std::vector<real_type> q(q_d.size());
    q_d.copy_to_host(q);

    // safe q vector for later use on the device
    q_d_ = std::move(q_d);

    return q;
}

template std::vector<float> csvm::generate_q2_impl(const ::plssvm::detail::parameter<float> &, const std::size_t, const std::size_t);
template std::vector<double> csvm::generate_q2_impl(const ::plssvm::detail::parameter<double> &, const std::size_t, const std::size_t);

template <typename real_type>
void csvm::assemble_kernel_matrix_explicit_impl(const ::plssvm::detail::parameter<real_type> &params, const std::size_t num_rows_reduced, const std::size_t num_features, [[maybe_unused]] const std::vector<real_type> &q_red, real_type QA_cost) {
    const dim3 block(32, 32);
    const dim3 grid(static_cast<int>(std::ceil(num_rows_reduced / static_cast<double>(block.x))),
                    static_cast<int>(std::ceil(num_rows_reduced / static_cast<double>(block.y))));

    const device_ptr_type<real_type> &q_d = std::get<device_ptr_type<real_type>>(q_d_);
    const device_ptr_type<real_type> &data_d = std::get<device_ptr_type<real_type>>(data_d_);

    device_ptr_type<real_type> kernel_matrix{ num_rows_reduced * num_rows_reduced };

    detail::set_device(0);
    switch (params.kernel_type) {
        case kernel_function_type::linear:
            cuda::device_kernel_assembly_linear<<<grid, block>>>(q_d.get(), kernel_matrix.get(), data_d.get(), QA_cost, real_type{ 1.0 } / params.cost, static_cast<kernel_index_type>(num_rows_reduced), static_cast<kernel_index_type>(num_features));
            break;
        case kernel_function_type::polynomial:
            cuda::device_kernel_assembly_polynomial<<<grid, block>>>(q_d.get(), kernel_matrix.get(), data_d.get(), QA_cost, real_type{ 1.0 } / params.cost, static_cast<kernel_index_type>(num_rows_reduced), static_cast<kernel_index_type>(num_features), params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_function_type::rbf:
            cuda::device_kernel_assembly_rbf<<<grid, block>>>(q_d.get(), kernel_matrix.get(), data_d.get(), QA_cost, real_type{ 1.0 } / params.cost, static_cast<kernel_index_type>(num_rows_reduced), static_cast<kernel_index_type>(num_features), params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
    detail::device_synchronize(0);

    // safe kernel matrix
    explicit_kernel_matrix_ = std::move(kernel_matrix);
}

template void csvm::assemble_kernel_matrix_explicit_impl(const ::plssvm::detail::parameter<float> &, const std::size_t, const std::size_t, const std::vector<float> &, float);
template void csvm::assemble_kernel_matrix_explicit_impl(const ::plssvm::detail::parameter<double> &, const std::size_t, const std::size_t, const std::vector<double> &, double);

template <typename real_type>
aos_matrix<real_type> csvm::kernel_matrix_matmul_explicit_impl(const aos_matrix<real_type> &vec) {
    const std::size_t num_rhs = vec.num_rows();
    const std::size_t num_rows = vec.num_cols();

    const device_ptr_type<real_type> &kernel_matrix = std::get<device_ptr_type<real_type>>(explicit_kernel_matrix_);
    device_ptr_type<real_type> vec_d{ vec.num_entries() };
    vec_d.copy_to_device(vec.data());

    device_ptr_type<real_type> ret_d{ vec.num_entries() };

    const dim3 block(32, 32);
    const dim3 grid(static_cast<int>(std::ceil(num_rows / static_cast<double>(block.x))),
                    static_cast<int>(std::ceil(num_rhs / static_cast<double>(block.y))));

    detail::set_device(0);
    cuda::device_kernel_gemm<<<grid, block>>>(num_rows, num_rhs, num_rows, real_type{ 1.0 }, kernel_matrix.get(), vec_d.get(), real_type{ 0.0 }, ret_d.get());  // TODO: easier
    detail::peek_at_last_error();
    detail::device_synchronize(0);

    aos_matrix<real_type> ret{ vec.num_rows(), vec.num_cols() };
    ret_d.copy_to_host(ret.data());
    return ret;
}

template aos_matrix<float> csvm::kernel_matrix_matmul_explicit_impl(const aos_matrix<float> &);
template aos_matrix<double> csvm::kernel_matrix_matmul_explicit_impl(const aos_matrix<double> &);

template <typename real_type>
void csvm::clear_data_on_devices_impl(real_type) {
    // clear device vectors!
    std::get<device_ptr_type<real_type>>(data_d_) = device_ptr_type<real_type>{};
    std::get<device_ptr_type<real_type>>(data_last_d_) = device_ptr_type<real_type>{};
    std::get<device_ptr_type<real_type>>(q_d_) = device_ptr_type<real_type>{};
    std::get<device_ptr_type<real_type>>(explicit_kernel_matrix_) = device_ptr_type<real_type>{};
}

template void csvm::clear_data_on_devices_impl(float);
template void csvm::clear_data_on_devices_impl(double);

}  // namespace plssvm::cuda
