#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/csvm.hpp"

#include "plssvm/backend_types.hpp"                                          // plssvm::backend_type
#include "plssvm/backends/CUDA/cg_explicit/blas.cuh"                         // plssvm::cuda::device_kernel_gemm
#include "plssvm/backends/CUDA/cg_explicit/kernel_matrix_assembly.cuh"       // plssvm::cuda::{device_kernel_assembly_linear, device_kernel_assembly_polynomial, device_kernel_assembly_rbf}
#include "plssvm/backends/CUDA/cg_implicit/kernel_matrix_assembly_blas.cuh"  // plssvm::cuda::device_kernel_assembly_linear_symm, plssvm::cuda::device_kernel_assembly_polynomial_symm, plssvm::cuda::device_kernel_assembly_rbf_symm
#include "plssvm/backends/CUDA/detail/device_ptr.cuh"                        // plssvm::cuda::detail::device_ptr
#include "plssvm/backends/CUDA/detail/utility.cuh"                           // plssvm::cuda::detail::{device_synchronize, get_device_count, set_device, peek_at_last_error, get_runtime_version}
#include "plssvm/backends/CUDA/exceptions.hpp"                               // plssvm::cuda::backend_exception
#include "plssvm/backends/CUDA/predict_kernel.cuh"                           // plssvm::cuda::detail::{device_kernel_w_linear, device_kernel_predict_polynomial, device_kernel_predict_rbf}
#include "plssvm/constants.hpp"                                              // plssvm::{real_type, THREAD_BLOCK_SIZE, INTERNAL_BLOCK_SIZE, PADDING_SIZE}
#include "plssvm/detail/assert.hpp"                                          // PLSSVM_ASSERT
#include "plssvm/detail/logging.hpp"                                         // plssvm::detail::log
#include "plssvm/detail/memory_size.hpp"                                     // plssvm::detail::memory_size
#include "plssvm/detail/performance_tracker.hpp"                             // plssvm::detail::tracking_entry
#include "plssvm/exceptions/exceptions.hpp"                                  // plssvm::exception
#include "plssvm/kernel_function_types.hpp"                                  // plssvm::kernel_function_type
#include "plssvm/parameter.hpp"                                              // plssvm::parameter
#include "plssvm/shape.hpp"                                                  // plssvm::shape
#include "plssvm/target_platforms.hpp"                                       // plssvm::target_platform
#include "plssvm/verbosity_levels.hpp"                                       // plssvm::verbosity_level

#include "hip/hip_runtime.h"              // cuda runtime functions
#include "hip/hip_runtime_api.h"  // cuda runtime functions

#include "fmt/color.h"    // fmt::fg, fmt::color::orange
#include "fmt/core.h"     // fmt::format
#include "fmt/ostream.h"  // can use fmt using operator<< overloads

#include <cmath>      // std::sqrt, std::ceil
#include <cstddef>    // std::size_t
#include <exception>  // std::terminate
#include <iostream>   // std::cout, std::endl
#include <numeric>    // std::iota
#include <string>     // std::string
#include <vector>     // std:vector

namespace plssvm::cuda {

csvm::csvm(parameter params) :
    csvm{ plssvm::target_platform::automatic, params } {}

csvm::csvm(target_platform target, parameter params) :
    base_type{ params } {
    this->init(target);
}

csvm::~csvm() {
    try {
        // be sure that all operations on the CUDA devices have finished before destruction
        for (const queue_type &device : devices_) {
            detail::device_synchronize(device);
        }
    } catch (const plssvm::exception &e) {
        std::cout << e.what_with_loc() << std::endl;
        std::terminate();
    }
}

void csvm::init(const target_platform target) {
    // check if supported target platform has been selected
    if (target != target_platform::automatic && target != target_platform::gpu_nvidia) {
        throw backend_exception{ fmt::format("Invalid target platform '{}' for the CUDA backend!", target) };
    } else {
#if !defined(PLSSVM_HAS_NVIDIA_TARGET)
        throw backend_exception{ "Requested target platform 'gpu_nvidia' that hasn't been enabled using PLSSVM_TARGET_PLATFORMS!" };
#endif
    }

    plssvm::detail::log(verbosity_level::full,
                        "\nUsing CUDA ({}) as backend.\n",
                        plssvm::detail::tracking_entry{ "dependencies", "cuda_runtime_version", detail::get_runtime_version() });
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "backend", plssvm::backend_type::cuda }));
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "target_platform", plssvm::target_platform::gpu_nvidia }));

    // update the target platform
    target_ = plssvm::target_platform::gpu_nvidia;

    // get all available devices wrt the requested target platform
    devices_.resize(detail::get_device_count());
    std::iota(devices_.begin(), devices_.end(), 0);

    // currently only single GPU execution is supported
    if (devices_.size() > 1) {
        plssvm::detail::log(verbosity_level::full | verbosity_level::warning,
                            "WARNING: found {} devices, but currently only single GPU execution is supported. Continuing only with device 0!\n",
                            devices_.size());
        devices_.resize(1);
    }

    // throw exception if no CUDA devices could be found
    if (devices_.empty()) {
        throw backend_exception{ "CUDA backend selected but no CUDA capable devices were found!" };
    }

    // print found CUDA devices
    plssvm::detail::log(verbosity_level::full,
                        "Found {} CUDA device(s):\n",
                        plssvm::detail::tracking_entry{ "backend", "num_devices", devices_.size() });
    std::vector<std::string> device_names;
    device_names.reserve(devices_.size());
    for (const queue_type &device : devices_) {
        hipDeviceProp_t prop{};
        hipGetDeviceProperties(&prop, device);
        plssvm::detail::log(verbosity_level::full,
                            "  [{}, {}, {}.{}]\n",
                            device,
                            prop.name,
                            prop.major,
                            prop.minor);
        device_names.emplace_back(prop.name);
    }
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "device", device_names }));
    plssvm::detail::log(verbosity_level::full | verbosity_level::timing,
                        "\n");
}

::plssvm::detail::memory_size csvm::get_device_memory() const {
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, devices_[0]);
    return ::plssvm::detail::memory_size{ static_cast<unsigned long long>(prop.totalGlobalMem) };
}

::plssvm::detail::memory_size csvm::get_max_mem_alloc_size() const {
    return this->get_device_memory();
}

std::size_t csvm::get_max_work_group_size() const {
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, devices_[0]);
    return static_cast<std::size_t>(prop.maxThreadsPerBlock);
}

//***************************************************//
//                        fit                        //
//***************************************************//

auto csvm::run_assemble_kernel_matrix_explicit(const parameter &params, const device_ptr_type &data_d, const device_ptr_type &q_red_d, real_type QA_cost) const -> device_ptr_type {
    const unsigned long long num_rows_reduced = data_d.shape().x - 1;
    const unsigned long long num_features = data_d.shape().y;

    // define grid and block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    if (max_work_group_size < THREAD_BLOCK_SIZE * THREAD_BLOCK_SIZE) {
        throw kernel_launch_resources{ fmt::format("Not enough work-items allowed for a work-groups of size {}x{}! Try reducing THREAD_BLOCK_SIZE.", THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE) };
    }
    const dim3 block(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_rows_reduced) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                    static_cast<int>(std::ceil(static_cast<double>(num_rows_reduced) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

#if defined(PLSSVM_USE_GEMM)
    device_ptr_type kernel_matrix_d{ (num_rows_reduced + PADDING_SIZE) * (num_rows_reduced + PADDING_SIZE), devices_[0] };  // store full matrix
#else
    device_ptr_type kernel_matrix_d{ (num_rows_reduced + PADDING_SIZE) * (num_rows_reduced + PADDING_SIZE + 1) / 2, devices_[0] };  // only explicitly store the upper triangular matrix
#endif
    kernel_matrix_d.memset(0);
    const real_type cost_factor = real_type{ 1.0 } / params.cost;

    detail::set_device(0);
    switch (params.kernel_type) {
        case kernel_function_type::linear:
            cuda::device_kernel_assembly_linear<<<grid, block>>>(kernel_matrix_d.get(), data_d.get(), num_rows_reduced, num_features, q_red_d.get(), QA_cost, cost_factor);
            break;
        case kernel_function_type::polynomial:
            cuda::device_kernel_assembly_polynomial<<<grid, block>>>(kernel_matrix_d.get(), data_d.get(), num_rows_reduced, num_features, q_red_d.get(), QA_cost, cost_factor, params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_function_type::rbf:
            cuda::device_kernel_assembly_rbf<<<grid, block>>>(kernel_matrix_d.get(), data_d.get(), num_rows_reduced, num_features, q_red_d.get(), QA_cost, cost_factor, params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
    detail::device_synchronize(devices_[0]);

    return kernel_matrix_d;
}

void csvm::run_blas_level_3_kernel_explicit(const real_type alpha, const device_ptr_type &A_d, const device_ptr_type &B_d, const real_type beta, device_ptr_type &C_d) const {
    const unsigned long long num_rhs = B_d.shape().x;
    const unsigned long long num_rows = B_d.shape().y;

    // define the grid and block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    if (max_work_group_size < THREAD_BLOCK_SIZE * THREAD_BLOCK_SIZE) {
        throw kernel_launch_resources{ fmt::format("Not enough work-items allowed for a work-groups of size {}x{}! Try reducing THREAD_BLOCK_SIZE.", THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE) };
    }
    const dim3 block(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_rhs) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                    static_cast<int>(std::ceil(static_cast<double>(num_rows) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

    detail::set_device(0);
#if defined(PLSSVM_USE_GEMM)
    cuda::device_kernel_gemm<<<grid, block>>>(num_rows, num_rhs, num_rows, alpha, A_d.get(), B_d.get(), beta, C_d.get());
#else
    cuda::device_kernel_symm<<<grid, block>>>(num_rows, num_rhs, num_rows, alpha, A_d.get(), B_d.get(), beta, C_d.get());
#endif
    detail::peek_at_last_error();
    detail::device_synchronize(devices_[0]);
}

void csvm::run_assemble_kernel_matrix_implicit_blas_level_3(const real_type alpha, const device_ptr_type &A_d, const parameter &params, const device_ptr_type &q_red, const real_type QA_cost, const device_ptr_type &B_d, device_ptr_type &C_d) const {
    const unsigned long long num_rows_reduced = A_d.shape().x - 1;
    const unsigned long long num_features = A_d.shape().y;
    const unsigned long long num_classes = B_d.shape().x;

    // define the grid and block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    if (max_work_group_size < THREAD_BLOCK_SIZE * THREAD_BLOCK_SIZE) {
        throw kernel_launch_resources{ fmt::format("Not enough work-items allowed for a work-groups of size {}x{}! Try reducing THREAD_BLOCK_SIZE.", THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE) };
    }
    const dim3 block(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_rows_reduced) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                    static_cast<int>(std::ceil(static_cast<double>(num_rows_reduced) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

    detail::set_device(0);
    const real_type cost_factor = real_type{ 1.0 } / params.cost;

    switch (params.kernel_type) {
        case kernel_function_type::linear:
            cuda::device_kernel_assembly_linear_symm<<<grid, block>>>(alpha, q_red.get(), A_d.get(), num_rows_reduced, num_features, QA_cost, cost_factor, B_d.get(), C_d.get(), num_classes);
            break;
        case kernel_function_type::polynomial:
            cuda::device_kernel_assembly_polynomial_symm<<<grid, block>>>(alpha, q_red.get(), A_d.get(), num_rows_reduced, num_features, QA_cost, cost_factor, params.degree.value(), params.gamma.value(), params.coef0.value(), B_d.get(), C_d.get(), num_classes);
            break;
        case kernel_function_type::rbf:
            cuda::device_kernel_assembly_rbf_symm<<<grid, block>>>(alpha, q_red.get(), A_d.get(), num_rows_reduced, num_features, QA_cost, cost_factor, params.gamma.value(), B_d.get(), C_d.get(), num_classes);
            break;
    }
    detail::peek_at_last_error();
    detail::device_synchronize(devices_[0]);
}

//***************************************************//
//                   predict, score                  //
//***************************************************//

auto csvm::run_w_kernel(const device_ptr_type &alpha_d, const device_ptr_type &sv_d) const -> device_ptr_type {
    const unsigned long long num_classes = alpha_d.shape().x;
    const unsigned long long num_sv = sv_d.shape().x;
    const unsigned long long num_features = sv_d.shape().y;

    // define the grid and block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    if (max_work_group_size < THREAD_BLOCK_SIZE * THREAD_BLOCK_SIZE) {
        throw kernel_launch_resources{ fmt::format("Not enough work-items allowed for a work-groups of size {}x{}! Try reducing THREAD_BLOCK_SIZE.", THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE) };
    }
    const dim3 block(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_features) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                    static_cast<int>(std::ceil(static_cast<double>(num_classes) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

    device_ptr_type w_d{ shape{ num_classes, num_features }, shape{ PADDING_SIZE, PADDING_SIZE }, devices_[0] };

    detail::set_device(0);
    cuda::device_kernel_w_linear<<<grid, block>>>(w_d.get(), alpha_d.get(), sv_d.get(), num_classes, num_sv);
    detail::peek_at_last_error();
    detail::device_synchronize(devices_[0]);

    return w_d;
}

auto csvm::run_predict_kernel(const parameter &params, const device_ptr_type &w_d, const device_ptr_type &alpha_d, const device_ptr_type &rho_d, const device_ptr_type &sv_d, const device_ptr_type &predict_points_d) const -> device_ptr_type {
    const unsigned long long num_classes = alpha_d.shape().x;
    const unsigned long long num_sv = sv_d.shape().x;
    const unsigned long long num_predict_points = predict_points_d.shape().x;
    const unsigned long long num_features = predict_points_d.shape().y;

    device_ptr_type out_d{ shape{ num_predict_points, num_classes }, shape{ PADDING_SIZE, PADDING_SIZE }, devices_[0] };

    // define the block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    if (max_work_group_size < THREAD_BLOCK_SIZE * THREAD_BLOCK_SIZE) {
        throw kernel_launch_resources{ fmt::format("Not enough work-items allowed for a work-groups of size {}x{}! Try reducing THREAD_BLOCK_SIZE.", THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE) };
    }
    const dim3 block(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);

    detail::set_device(0);
    if (params.kernel_type == kernel_function_type::linear) {
        // define the grid sizes
        const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_predict_points) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                        static_cast<int>(std::ceil(static_cast<double>(num_classes) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

        cuda::device_kernel_predict_linear<<<grid, block>>>(out_d.get(), w_d.get(), rho_d.get(), predict_points_d.get(), num_classes, num_predict_points, num_features);
    } else {
        // define the grid sizes
        const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_predict_points) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                        static_cast<int>(std::ceil(static_cast<double>(num_sv) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

        switch (params.kernel_type) {
            case kernel_function_type::linear:
                // already handled
                break;
            case kernel_function_type::polynomial:
                cuda::device_kernel_predict_polynomial<<<grid, block>>>(out_d.get(), alpha_d.get(), rho_d.get(), sv_d.get(), predict_points_d.get(), num_classes, num_sv, num_predict_points, num_features, params.degree.value(), params.gamma.value(), params.coef0.value());
                break;
            case kernel_function_type::rbf:
                cuda::device_kernel_predict_rbf<<<grid, block>>>(out_d.get(), alpha_d.get(), rho_d.get(), sv_d.get(), predict_points_d.get(), num_classes, num_sv, num_predict_points, num_features, params.gamma.value());
                break;
        }
    }
    detail::peek_at_last_error();
    detail::device_synchronize(devices_[0]);

    return out_d;
}

}  // namespace plssvm::cuda
