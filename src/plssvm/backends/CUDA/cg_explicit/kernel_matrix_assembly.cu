#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/cg_explicit/kernel_matrix_assembly.cuh"

#include "plssvm/constants.hpp"  // plssvm::real_type

namespace plssvm::cuda {

__global__ void device_kernel_assembly_linear(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < num_rows && j < num_rows && j >= i) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; ++dim) {
            temp += data_d[i * num_features + dim] * data_d[j * num_features + dim];
        }
        temp = temp + QA_cost - q[i] - q[j];
        if (i == j) {
            temp += cost;
        }

        ret[i * num_rows + j] = temp;
        ret[j * num_rows + i] = temp;
    }
}

__global__ void device_kernel_assembly_polynomial(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost, const int degree, const real_type gamma, const real_type coef0) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < num_rows && j < num_rows && j >= i) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; ++dim) {
            temp += data_d[i * num_features + dim] * data_d[j * num_features + dim];
        }
        temp = pow(gamma * temp + coef0, degree) + QA_cost - q[i] - q[j];
        if (i == j) {
            temp += cost;
        }

        ret[i * num_rows + j] = temp;
        ret[j * num_rows + i] = temp;
    }
}

__global__ void device_kernel_assembly_rbf(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost, const real_type gamma) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < num_rows && j < num_rows && j >= i) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; ++dim) {
            const real_type d = data_d[i * num_features + dim] - data_d[j * num_features + dim];
            temp += d * d;
        }
        temp = exp(-gamma * temp) + QA_cost - q[i] - q[j];
        if (i == j) {
            temp += cost;
        }

        ret[i * num_rows + j] = temp;
        ret[j * num_rows + i] = temp;
    }
}

}  // namespace plssvm::cuda