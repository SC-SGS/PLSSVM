#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/predict_kernel.cuh"

#include "plssvm/backends/CUDA/detail/atomics.cuh"  // atomicAdd for double precision floating point numbers on older CUDA hardware
#include "plssvm/constants.hpp"                     // plssvm::kernel_index_type, plssvm::THREAD_BLOCK_SIZE, plssvm::INTERNAL_BLOCK_SIZE

namespace plssvm::cuda {

template <typename real_type>
__global__ void device_kernel_w_linear(real_type *w_d, const real_type *data_d, const real_type *data_last_d, const real_type *alpha_d, const kernel_index_type num_data_points, const kernel_index_type num_features) {
    const kernel_index_type index = blockIdx.x * blockDim.x + threadIdx.x;
    real_type temp{ 0.0 };
    if (index < num_features) {
        for (kernel_index_type dat = 0; dat < num_data_points - 1; ++dat) {
            temp += alpha_d[dat] * data_d[dat + (num_data_points - 1 + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * index];
        }
        temp += alpha_d[num_data_points - 1] * data_last_d[index];
        w_d[index] = temp;
    }
}
template __global__ void device_kernel_w_linear(float *, const float *, const float *, const float *, const kernel_index_type, const kernel_index_type);
template __global__ void device_kernel_w_linear(double *, const double *, const double *, const double *, const kernel_index_type, const kernel_index_type);

template <typename real_type>
__global__ void device_kernel_predict_poly(real_type *out_d, const real_type *data_d, const real_type *data_last_d, const real_type *alpha_d, const kernel_index_type num_data_points, const real_type *points, const kernel_index_type num_predict_points, const kernel_index_type num_features, const int degree, const real_type gamma, const real_type coef0) {
    const kernel_index_type data_point_index = blockIdx.x * blockDim.x + threadIdx.x;
    const kernel_index_type predict_point_index = blockIdx.y * blockDim.y + threadIdx.y;

    real_type temp{ 0.0 };
    if (predict_point_index < num_predict_points) {
        for (kernel_index_type feature_index = 0; feature_index < num_features; ++feature_index) {
            if (data_point_index == num_data_points - 1) {
                temp += data_last_d[feature_index] * points[predict_point_index + (num_predict_points + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index];
            } else {
                temp += data_d[data_point_index + (num_data_points - 1 + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index] * points[predict_point_index + (num_predict_points + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index];
            }
        }

        temp = alpha_d[data_point_index] * pow(gamma * temp + coef0, degree);

        atomicAdd(&out_d[predict_point_index], temp);
    }
}

template __global__ void device_kernel_predict_poly(float *, const float *, const float *, const float *, const kernel_index_type, const float *, const kernel_index_type, const kernel_index_type, const int, const float, const float);
template __global__ void device_kernel_predict_poly(double *, const double *, const double *, const double *, const kernel_index_type, const double *, const kernel_index_type, const kernel_index_type, const int, const double, const double);

template <typename real_type>
__global__ void device_kernel_predict_radial(real_type *out_d, const real_type *data_d, const real_type *data_last_d, const real_type *alpha_d, const kernel_index_type num_data_points, const real_type *points, const kernel_index_type num_predict_points, const kernel_index_type num_features, const real_type gamma) {
    const kernel_index_type data_point_index = blockIdx.x * blockDim.x + threadIdx.x;
    const kernel_index_type predict_point_index = blockIdx.y * blockDim.y + threadIdx.y;

    real_type temp{ 0.0 };
    if (predict_point_index < num_predict_points) {
        for (kernel_index_type feature_index = 0; feature_index < num_features; ++feature_index) {
            if (data_point_index == num_data_points - 1) {
                temp += (data_last_d[feature_index] - points[predict_point_index + (num_predict_points + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index]) * (data_last_d[feature_index] - points[predict_point_index + (num_predict_points + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index]);
            } else {
                temp += (data_d[data_point_index + (num_data_points - 1 + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index] - points[predict_point_index + (num_predict_points + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index]) * (data_d[data_point_index + (num_data_points - 1 + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index] - points[predict_point_index + (num_predict_points + THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE) * feature_index]);
            }
        }

        temp = alpha_d[data_point_index] * exp(-gamma * temp);

        atomicAdd(&out_d[predict_point_index], temp);
    }
}

template __global__ void device_kernel_predict_radial(float *, const float *, const float *, const float *, const kernel_index_type, const float *, const kernel_index_type, const kernel_index_type, const float);
template __global__ void device_kernel_predict_radial(double *, const double *, const double *, const double *, const kernel_index_type, const double *, const kernel_index_type, const kernel_index_type, const double);

}  // namespace plssvm::cuda
