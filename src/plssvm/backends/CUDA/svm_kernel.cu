#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/svm_kernel.cuh"

#include "plssvm/backends/CUDA/detail/atomics.cuh"  // atomicAdd for double precision floating point numbers on older CUDA hardware
#include "plssvm/constants.hpp"                     // plssvm::THREAD_BLOCK_SIZE, plssvm::INTERNAL_BLOCK_SIZE

namespace plssvm::cuda {

template <typename real_type>
__global__ void device_kernel_linear(const real_type *q, real_type *ret, const real_type *d, const real_type *data_d, const real_type QA_cost, const real_type cost, const kernel_index_type num_rows, const kernel_index_type feature_range, const real_type add, const kernel_index_type id) {
    kernel_index_type i = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE;
    kernel_index_type j = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE;

    __shared__ real_type data_intern_i[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    __shared__ real_type data_intern_j[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    real_type matr[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };
    real_type data_j[INTERNAL_BLOCK_SIZE];

    if (i >= j) {
        i += threadIdx.x * INTERNAL_BLOCK_SIZE;
        const kernel_index_type ji = j + threadIdx.x * INTERNAL_BLOCK_SIZE;
        j += threadIdx.y * INTERNAL_BLOCK_SIZE;
        // cache data
        for (kernel_index_type vec_index = 0; vec_index < feature_range * num_rows; vec_index += num_rows) {
            __syncthreads();
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type block_id = 0; block_id < INTERNAL_BLOCK_SIZE; ++block_id) {
                const kernel_index_type idx = block_id % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx) {
                    data_intern_i[threadIdx.x][block_id] = data_d[block_id + vec_index + i];
                }
                const kernel_index_type idx_2 = block_id + INTERNAL_BLOCK_SIZE % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx_2) {
                    data_intern_j[threadIdx.x][block_id] = data_d[block_id + vec_index + ji];
                }
            }
            __syncthreads();

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type data_index = 0; data_index < INTERNAL_BLOCK_SIZE; ++data_index) {
                data_j[data_index] = data_intern_j[threadIdx.y][data_index];
            }

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type l = 0; l < INTERNAL_BLOCK_SIZE; ++l) {
                const real_type data_i = data_intern_i[threadIdx.x][l];
                #pragma unroll INTERNAL_BLOCK_SIZE
                for (kernel_index_type k = 0; k < INTERNAL_BLOCK_SIZE; ++k) {
                    matr[k][l] += data_i * data_j[k];
                }
            }
        }

        #pragma unroll INTERNAL_BLOCK_SIZE
        for (kernel_index_type x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            real_type ret_jx = 0.0;
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                real_type temp;
                if (id == 0) {
                    temp = (matr[x][y] + QA_cost - q[i + y] - q[j + x]) * add;
                } else {
                    temp = matr[x][y] * add;
                }
                if (i + x > j + y) {
                    // upper triangular matrix
                    atomicAdd(&ret[i + y], temp * d[j + x]);
                    ret_jx += temp * d[i + y];
                } else if (i + x == j + y) {
                    // diagonal
                    if (id == 0) {
                        ret_jx += (temp + cost * add) * d[i + y];
                    } else {
                        ret_jx += temp * d[i + y];
                    }
                }
            }
            atomicAdd(&ret[j + x], ret_jx);
        }
    }
}

template __global__ void device_kernel_linear(const float *, float *, const float *, const float *, const float, const float, const kernel_index_type, const kernel_index_type, const float, const kernel_index_type);
template __global__ void device_kernel_linear(const double *, double *, const double *, const double *, const double, const double, const kernel_index_type, const kernel_index_type, const double, const kernel_index_type);

template <typename real_type>
__global__ void device_kernel_polynomial(const real_type *q, real_type *ret, const real_type *d, const real_type *data_d, const real_type QA_cost, const real_type cost, const kernel_index_type num_rows, const kernel_index_type num_cols, const real_type add, const int degree, const real_type gamma, const real_type coef0) {
    kernel_index_type i = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE;
    kernel_index_type j = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE;

    __shared__ real_type data_intern_i[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    __shared__ real_type data_intern_j[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    real_type matr[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };
    real_type data_j[INTERNAL_BLOCK_SIZE];

    if (i >= j) {
        i += threadIdx.x * INTERNAL_BLOCK_SIZE;
        const kernel_index_type ji = j + threadIdx.x * INTERNAL_BLOCK_SIZE;
        j += threadIdx.y * INTERNAL_BLOCK_SIZE;
        for (kernel_index_type vec_index = 0; vec_index < num_cols * num_rows; vec_index += num_rows) {
            __syncthreads();
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type block_id = 0; block_id < INTERNAL_BLOCK_SIZE; ++block_id) {
                const kernel_index_type idx = block_id % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx) {
                    data_intern_i[threadIdx.x][block_id] = data_d[block_id + vec_index + i];
                }
                const kernel_index_type idx_2 = block_id + INTERNAL_BLOCK_SIZE % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx_2) {
                    data_intern_j[threadIdx.x][block_id] = data_d[block_id + vec_index + ji];
                }
            }
            __syncthreads();

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type data_index = 0; data_index < INTERNAL_BLOCK_SIZE; ++data_index) {
                data_j[data_index] = data_intern_j[threadIdx.y][data_index];
            }

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type l = 0; l < INTERNAL_BLOCK_SIZE; ++l) {
                const real_type data_i = data_intern_i[threadIdx.x][l];
                #pragma unroll INTERNAL_BLOCK_SIZE
                for (kernel_index_type k = 0; k < INTERNAL_BLOCK_SIZE; ++k) {
                    matr[k][l] += data_i * data_j[k];
                }
            }
        }

        #pragma unroll INTERNAL_BLOCK_SIZE
        for (kernel_index_type x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            real_type ret_jx = 0.0;
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                const real_type temp = (pow(gamma * matr[x][y] + coef0, degree) + QA_cost - q[i + y] - q[j + x]) * add;
                if (i + x > j + y) {
                    // upper triangular matrix
                    atomicAdd(&ret[i + y], temp * d[j + x]);
                    ret_jx += temp * d[i + y];
                } else if (i + x == j + y) {
                    // diagonal
                    ret_jx += (temp + cost * add) * d[i + y];
                }
            }
            atomicAdd(&ret[j + x], ret_jx);
        }
    }
}

template __global__ void device_kernel_polynomial(const float *, float *, const float *, const float *, const float, const float, const kernel_index_type, const kernel_index_type, const float, const int, const float, const float);
template __global__ void device_kernel_polynomial(const double *, double *, const double *, const double *, const double, const double, const kernel_index_type, const kernel_index_type, const double, const int, const double, const double);

template <typename real_type>
__global__ void device_kernel_rbf(const real_type *q, real_type *ret, const real_type *d, const real_type *data_d, const real_type QA_cost, const real_type cost, const kernel_index_type num_rows, const kernel_index_type num_cols, const real_type add, const real_type gamma) {
    kernel_index_type i = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE;
    kernel_index_type j = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE;

    __shared__ real_type data_intern_i[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    __shared__ real_type data_intern_j[THREAD_BLOCK_SIZE][INTERNAL_BLOCK_SIZE];
    real_type matr[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };
    real_type data_j[INTERNAL_BLOCK_SIZE];

    if (i >= j) {
        i += threadIdx.x * INTERNAL_BLOCK_SIZE;
        const kernel_index_type ji = j + threadIdx.x * INTERNAL_BLOCK_SIZE;
        j += threadIdx.y * INTERNAL_BLOCK_SIZE;
        for (kernel_index_type vec_index = 0; vec_index < num_cols * num_rows; vec_index += num_rows) {
            __syncthreads();
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type block_id = 0; block_id < INTERNAL_BLOCK_SIZE; ++block_id) {
                const kernel_index_type idx = block_id % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx) {
                    data_intern_i[threadIdx.x][block_id] = data_d[block_id + vec_index + i];
                }
                const kernel_index_type idx2 = block_id + INTERNAL_BLOCK_SIZE % THREAD_BLOCK_SIZE;
                if (threadIdx.y == idx2) {
                    data_intern_j[threadIdx.x][block_id] = data_d[block_id + vec_index + ji];
                }
            }
            __syncthreads();

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type data_index = 0; data_index < INTERNAL_BLOCK_SIZE; ++data_index) {
                data_j[data_index] = data_intern_j[threadIdx.y][data_index];
            }

            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type l = 0; l < INTERNAL_BLOCK_SIZE; ++l) {
                const real_type data_i = data_intern_i[threadIdx.x][l];
                #pragma unroll INTERNAL_BLOCK_SIZE
                for (kernel_index_type k = 0; k < INTERNAL_BLOCK_SIZE; ++k) {
                    matr[k][l] += (data_i - data_j[k]) * (data_i - data_j[k]);
                }
            }
        }

        #pragma unroll INTERNAL_BLOCK_SIZE
        for (kernel_index_type x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            real_type ret_jx = 0.0;
            #pragma unroll INTERNAL_BLOCK_SIZE
            for (kernel_index_type y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
                const real_type temp = (exp(-gamma * matr[x][y]) + QA_cost - q[i + y] - q[j + x]) * add;
                if (i + x > j + y) {
                    // upper triangular matrix
                    atomicAdd(&ret[i + y], temp * d[j + x]);
                    ret_jx += temp * d[i + y];
                } else if (i + x == j + y) {
                    // diagonal
                    ret_jx += (temp + cost * add) * d[i + y];
                }
            }
            atomicAdd(&ret[j + x], ret_jx);
        }
    }
}
template __global__ void device_kernel_rbf(const float *, float *, const float *, const float *, const float, const float, const kernel_index_type, const kernel_index_type, const float, const float);
template __global__ void device_kernel_rbf(const double *, double *, const double *, const double *, const double, const double, const kernel_index_type, const kernel_index_type, const double, const double);

}  // namespace plssvm::cuda