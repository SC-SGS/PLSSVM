#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/cg_explicit/kernel_matrix_assembly.cuh"

#include "plssvm/constants.hpp"  // plssvm::real_type, plssvm::THREAD_BLOCK_SIZE, plssvm::FEATURE_BLOCK_SIZE

namespace plssvm::cuda {

__global__ void device_kernel_assembly_linear(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned long long j_cached_idx = blockIdx.y * blockDim.y + threadIdx.x;

    __shared__ real_type data_cache_i[FEATURE_BLOCK_SIZE][THREAD_BLOCK_SIZE];
    __shared__ real_type data_cache_j[FEATURE_BLOCK_SIZE][THREAD_BLOCK_SIZE];

    if (blockIdx.x >= blockIdx.y) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; dim += FEATURE_BLOCK_SIZE) {
            // zero out shared memory
            if (threadIdx.y < FEATURE_BLOCK_SIZE) {
                data_cache_i[threadIdx.y][threadIdx.x] = real_type{ 0.0 };
                data_cache_j[threadIdx.y][threadIdx.x] = real_type{ 0.0 };
            }

            // load data into shared memory
            if (threadIdx.y < FEATURE_BLOCK_SIZE && dim + threadIdx.y < num_features) {
                if (i < num_rows) {
                    data_cache_i[threadIdx.y][threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + i];
                }
                if (j_cached_idx < num_rows) {
                    data_cache_j[threadIdx.y][threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + j_cached_idx];
                }
            }
            __syncthreads();

            // calculation
            for (unsigned long long block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
                temp += data_cache_i[block_dim][threadIdx.x] * data_cache_j[block_dim][threadIdx.y];
            }
            __syncthreads();
        }

        if (i < num_rows && j < num_rows && i >= j) {
            temp = temp + QA_cost - q[i] - q[j];
            if (i == j) {
                temp += cost;
            }

#if defined(PLSSVM_USE_GEMM)
            ret[j * num_rows + i] = temp;
            ret[i * num_rows + j] = temp;
#else
            ret[j * num_rows + i - j * (j + 1) / 2] = temp;
#endif
        }
    }
}

__global__ void device_kernel_assembly_polynomial(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost, const int degree, const real_type gamma, const real_type coef0) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned long long j_cached_idx = blockIdx.y * blockDim.y + threadIdx.x;

    __shared__ real_type data_cache_i[FEATURE_BLOCK_SIZE][THREAD_BLOCK_SIZE];
    __shared__ real_type data_cache_j[FEATURE_BLOCK_SIZE][THREAD_BLOCK_SIZE];

    if (blockIdx.x >= blockIdx.y) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; dim += FEATURE_BLOCK_SIZE) {
            // zero out shared memory
            if (threadIdx.y < FEATURE_BLOCK_SIZE) {
                data_cache_i[threadIdx.y][threadIdx.x] = real_type{ 0.0 };
                data_cache_j[threadIdx.y][threadIdx.x] = real_type{ 0.0 };
            }

            // load data into shared memory
            if (threadIdx.y < FEATURE_BLOCK_SIZE && dim + threadIdx.y < num_features) {
                if (i < num_rows) {
                    data_cache_i[threadIdx.y][threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + i];
                }
                if (j_cached_idx < num_rows) {
                    data_cache_j[threadIdx.y][threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + j_cached_idx];
                }
            }
            __syncthreads();

            // calculation
            for (unsigned long long block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
                temp += data_cache_i[block_dim][threadIdx.x] * data_cache_j[block_dim][threadIdx.y];
            }
            __syncthreads();
        }

        if (i < num_rows && j < num_rows && i >= j) {
            temp = pow(gamma * temp + coef0, (double) degree) + QA_cost - q[i] - q[j];
            if (i == j) {
                temp += cost;
            }

#if defined(PLSSVM_USE_GEMM)
            ret[j * num_rows + i] = temp;
            ret[i * num_rows + j] = temp;
#else
            ret[j * num_rows + i - j * (j + 1) / 2] = temp;
#endif
        }
    }
}

__global__ void device_kernel_assembly_rbf(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost, const real_type gamma) {
    const unsigned long long i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long long j = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned long long j_cached_idx = blockIdx.y * blockDim.y + threadIdx.x;

    __shared__ real_type data_cache_i[FEATURE_BLOCK_SIZE][THREAD_BLOCK_SIZE];
    __shared__ real_type data_cache_j[FEATURE_BLOCK_SIZE][THREAD_BLOCK_SIZE];

    if (blockIdx.x >= blockIdx.y) {
        real_type temp{ 0.0 };
        for (unsigned long long dim = 0; dim < num_features; dim += FEATURE_BLOCK_SIZE) {
            // zero out shared memory
            if (threadIdx.y < FEATURE_BLOCK_SIZE) {
                data_cache_i[threadIdx.y][threadIdx.x] = real_type{ 0.0 };
                data_cache_j[threadIdx.y][threadIdx.x] = real_type{ 0.0 };
            }

            // load data into shared memory
            if (threadIdx.y < FEATURE_BLOCK_SIZE && dim + threadIdx.y < num_features) {
                if (i < num_rows) {
                    data_cache_i[threadIdx.y][threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + i];
                }
                if (j_cached_idx < num_rows) {
                    data_cache_j[threadIdx.y][threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + j_cached_idx];
                }
            }
            __syncthreads();

            // calculation
            for (unsigned long long block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
                const real_type d = data_cache_i[block_dim][threadIdx.x] - data_cache_j[block_dim][threadIdx.y];
                temp += d * d;
            }
            __syncthreads();
        }

        if (i < num_rows && j < num_rows && i >= j) {
            temp = exp(-gamma * temp) + QA_cost - q[i] - q[j];
            if (i == j) {
                temp += cost;
            }

#if defined(PLSSVM_USE_GEMM)
            ret[j * num_rows + i] = temp;
            ret[i * num_rows + j] = temp;
#else
            ret[j * num_rows + i - j * (j + 1) / 2] = temp;
#endif
        }
    }
}

}  // namespace plssvm::cuda