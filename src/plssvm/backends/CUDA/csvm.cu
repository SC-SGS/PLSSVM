/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/csvm.hpp"

#include "plssvm/backends/CUDA/detail/device_ptr.cuh"  // plssvm::cuda::detail::device_ptr
#include "plssvm/backends/CUDA/detail/utility.cuh"     // plssvm::cuda::detail::device_synchronize, plssvm::detail::cuda::get_device_count, plssvm::detail::cuda::set_device, plssvm::detail::cuda::peek_at_last_error
#include "plssvm/backends/CUDA/exceptions.hpp"         // plssvm::cuda::backend_exception
#include "plssvm/backends/CUDA/predict_kernel.cuh"     // plssvm::cuda::kernel_w, plssvm::cuda::predict_points_poly, plssvm::cuda::predict_points_rbf
#include "plssvm/backends/CUDA/q_kernel.cuh"           // plssvm::cuda::device_kernel_q_linear, plssvm::cuda::device_kernel_q_poly, plssvm::cuda::device_kernel_q_radial
#include "plssvm/backends/CUDA/svm_kernel.cuh"         // plssvm::cuda::device_kernel_linear, plssvm::cuda::device_kernel_poly, plssvm::cuda::device_kernel_radial
#include "plssvm/backends/gpu_csvm.hpp"                // plssvm::detail::gpu_csvm
#include "plssvm/detail/assert.hpp"                    // PLSSVM_ASSERT
#include "plssvm/detail/execution_range.hpp"           // plssvm::detail::execution_range
#include "plssvm/exceptions/exceptions.hpp"            // plssvm::exception
#include "plssvm/kernel_types.hpp"                     // plssvm::kernel_type
#include "plssvm/parameter.hpp"                        // plssvm::parameter
#include "plssvm/target_platforms.hpp"                 // plssvm::target_platform

#include "fmt/core.h"     // fmt::print, fmt::format
#include "fmt/ostream.h"  // can use fmt using operator<< overloads

#include <exception>  // std::terminate
#include <numeric>    // std::iota
#include <utility>    // std::pair, std::make_pair
#include <vector>     // std::vector

namespace plssvm::cuda {

template <typename T>
csvm<T>::csvm(target_platform target, parameter<real_type> params) : base_type{ params } {
    this->init(target);
}

template <typename T>
void csvm<T>::init(const target_platform target) {
    // check if supported target platform has been selected
    if (target != target_platform::automatic && target != target_platform::gpu_nvidia) {
        throw backend_exception{ fmt::format("Invalid target platform '{}' for the CUDA backend!", target) };
    } else {
#if !defined(PLSSVM_HAS_NVIDIA_TARGET)
        throw backend_exception{ fmt::format("Requested target platform {} that hasn't been enabled using PLSSVM_TARGET_PLATFORMS!", target) };
#endif
    }

    if (plssvm::verbose) {
        fmt::print("Using CUDA as backend.\n");
    }

    // get all available devices wrt the requested target platform
    devices_.resize(detail::get_device_count());
    std::iota(devices_.begin(), devices_.end(), 0);

    // throw exception if no CUDA devices could be found
    if (devices_.empty()) {
        throw backend_exception{ "CUDA backend selected but no CUDA devices were found!" };
    }

    if (plssvm::verbose) {
        // print found CUDA devices
        fmt::print("Found {} CUDA device(s):\n", devices_.size());
        for (typename std::vector<queue_type>::size_type device = 0; device < devices_.size(); ++device) {
            hipDeviceProp_t prop{};
            hipGetDeviceProperties(&prop, devices_[device]);
            fmt::print("  [{}, {}, {}.{}]\n", devices_[device], prop.name, prop.major, prop.minor);
        }
        fmt::print("\n");
    }
}

template <typename T>
csvm<T>::~csvm() {
    try {
        // be sure that all operations on the CUDA devices have finished before destruction
        for (const queue_type &device : devices_) {
            detail::device_synchronize(device);
        }
    } catch (const plssvm::exception &e) {
        fmt::print("{}\n", e.what_with_loc());
        std::terminate();
    }
}

template <typename T>
void csvm<T>::device_synchronize(const queue_type &queue) const {
    detail::device_synchronize(queue);
}

std::pair<dim3, dim3> execution_range_to_native(const ::plssvm::detail::execution_range &range) {
    dim3 grid(range.grid[0], range.grid[1], range.grid[2]);
    dim3 block(range.block[0], range.block[1], range.block[2]);
    return std::make_pair(grid, block);
}

template <typename T>
void csvm<T>::run_q_kernel(const size_type device, const ::plssvm::detail::execution_range &range, const parameter<real_type> &params, device_ptr_type &q_d, const device_ptr_type &data_d, const device_ptr_type &data_last_d, const size_type num_data_points_padded, const size_type num_features) const {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    switch (params.kernel) {
        case kernel_type::linear:
            cuda::device_kernel_q_linear<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features));
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_poly<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_radial<<<grid, block>>>(q_d.get(), data_d.get(), data_last_d.get(), static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
}

template <typename T>
void csvm<T>::run_svm_kernel(const size_type device, const ::plssvm::detail::execution_range &range, const parameter<real_type> &params, const device_ptr_type &q_d, device_ptr_type &r_d, const device_ptr_type &x_d, const device_ptr_type &data_d, const real_type QA_cost, const real_type add, const size_type num_data_points_padded, const size_type num_features) const {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    switch (params.kernel) {
        case kernel_type::linear:
            cuda::device_kernel_linear<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d.get(), QA_cost, 1 / params.cost, static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), add, static_cast<kernel_index_type>(device));
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_poly<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d.get(), QA_cost, 1 / params.cost, static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), add, params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_radial<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d.get(), QA_cost, 1 / params.cost, static_cast<kernel_index_type>(num_data_points_padded), static_cast<kernel_index_type>(num_features), add, params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
}

template <typename T>
void csvm<T>::run_w_kernel(const size_type device, const ::plssvm::detail::execution_range &range, device_ptr_type &w_d, const device_ptr_type &alpha_d, const device_ptr_type &data_d, const device_ptr_type &data_last_d, const size_type num_data_points, const size_type num_features) const {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    cuda::device_kernel_w_linear<<<grid, block>>>(w_d.get(), data_d.get(), data_last_d.get(), alpha_d.get(), static_cast<kernel_index_type>(num_data_points), static_cast<kernel_index_type>(num_features));
    detail::peek_at_last_error();
}

template <typename T>
void csvm<T>::run_predict_kernel(const ::plssvm::detail::execution_range &range, const parameter<real_type> &params, device_ptr_type &out_d, const device_ptr_type &alpha_d, const device_ptr_type &point_d, const device_ptr_type &data_d, const device_ptr_type &data_last_d, const size_type num_support_vectors, const size_type num_predict_points, const size_type num_features) const {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(0);
    switch (params.kernel) {
        case kernel_type::linear:
            break;
        case kernel_type::polynomial:
            cuda::device_kernel_predict_poly<<<grid, block>>>(out_d.get(), data_d.get(), data_last_d.get(), alpha_d.get(), static_cast<kernel_index_type>(num_support_vectors), point_d.get(), static_cast<kernel_index_type>(num_predict_points), static_cast<kernel_index_type>(num_features), params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_type::rbf:
            cuda::device_kernel_predict_radial<<<grid, block>>>(out_d.get(), data_d.get(), data_last_d.get(), alpha_d.get(), static_cast<kernel_index_type>(num_support_vectors), point_d.get(), static_cast<kernel_index_type>(num_predict_points), static_cast<kernel_index_type>(num_features), params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
}

template class csvm<float>;
template class csvm<double>;

}  // namespace plssvm::cuda
