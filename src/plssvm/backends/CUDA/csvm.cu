#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/csvm.hpp"

#include "plssvm/backend_types.hpp"                                     // plssvm::backend_type
#include "plssvm/backends/CUDA/cg_explicit/blas.cuh"                    // plssvm::cuda::device_kernel_gemm
#include "plssvm/backends/CUDA/cg_explicit/kernel_matrix_assembly.cuh"  // plssvm::cuda::{device_kernel_assembly_linear, device_kernel_assembly_polynomial, device_kernel_assembly_rbf}
#include "plssvm/backends/CUDA/detail/device_ptr.cuh"                   // plssvm::cuda::detail::device_ptr
#include "plssvm/backends/CUDA/detail/utility.cuh"                      // plssvm::cuda::detail::{device_synchronize, get_device_count, set_device, peek_at_last_error}
#include "plssvm/backends/CUDA/exceptions.hpp"                          // plssvm::cuda::backend_exception
#include "plssvm/backends/CUDA/predict_kernel.cuh"                      // plssvm::cuda::detail::{device_kernel_w_linear, device_kernel_predict_polynomial, device_kernel_predict_rbf}
#include "plssvm/constants.hpp"                                         // plssvm::real_type
#include "plssvm/detail/assert.hpp"                                     // PLSSVM_ASSERT
#include "plssvm/detail/logger.hpp"                                     // plssvm::detail::log, plssvm::verbosity_level
#include "plssvm/detail/performance_tracker.hpp"                        // plssvm::detail::tracking_entry
#include "plssvm/exceptions/exceptions.hpp"                             // plssvm::exception
#include "plssvm/kernel_function_types.hpp"                             // plssvm::kernel_function_type
#include "plssvm/parameter.hpp"                                         // plssvm::parameter
#include "plssvm/target_platforms.hpp"                                  // plssvm::target_platform

#include "hip/hip_runtime.h"                                      // cuda runtime functions
#include "hip/hip_runtime_api.h"                          // cuda runtime functions

#include "fmt/core.h"                                  // fmt::format
#include "fmt/ostream.h"                               // can use fmt using operator<< overloads

#include <cstddef>                                     // std::size_t
#include <exception>                                   // std::terminate
#include <iostream>                                    // std::cout, std::endl
#include <numeric>                                     // std::iota
#include <utility>                                     // std::pair, std::make_pair

namespace plssvm::cuda {

csvm::csvm(parameter params) :
    csvm{ plssvm::target_platform::automatic, params } {}

csvm::csvm(target_platform target, parameter params) :
    base_type{ params } {
    this->init(target);
}

csvm::~csvm() {
    try {
        // be sure that all operations on the CUDA devices have finished before destruction
        for (const queue_type &device : devices_) {
            detail::device_synchronize(device);
        }
    } catch (const plssvm::exception &e) {
        std::cout << e.what_with_loc() << std::endl;
        std::terminate();
    }
}

void csvm::init(const target_platform target) {
    // check if supported target platform has been selected
    if (target != target_platform::automatic && target != target_platform::gpu_nvidia) {
        throw backend_exception{ fmt::format("Invalid target platform '{}' for the CUDA backend!", target) };
    } else {
#if !defined(PLSSVM_HAS_NVIDIA_TARGET)
        throw backend_exception{ "Requested target platform 'gpu_nvidia' that hasn't been enabled using PLSSVM_TARGET_PLATFORMS!" };
#endif
    }

    plssvm::detail::log(verbosity_level::full,
                        "\nUsing CUDA as backend.\n");
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "backend", plssvm::backend_type::cuda }));
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "target_platform", plssvm::target_platform::gpu_nvidia }));

    // update the target platform
    target_ = plssvm::target_platform::gpu_nvidia;

    // get all available devices wrt the requested target platform
    devices_.resize(detail::get_device_count());
    std::iota(devices_.begin(), devices_.end(), 0);

    // throw exception if no CUDA devices could be found
    if (devices_.empty()) {
        throw backend_exception{ "CUDA backend selected but no CUDA capable devices were found!" };
    }

    // print found CUDA devices
    plssvm::detail::log(verbosity_level::full,
                        "Found {} CUDA device(s):\n", plssvm::detail::tracking_entry{ "backend", "num_devices", devices_.size() });
    std::vector<std::string> device_names;
    device_names.reserve(devices_.size());
    for (const queue_type &device : devices_) {
        hipDeviceProp_t prop{};
        hipGetDeviceProperties(&prop, device);
        plssvm::detail::log(verbosity_level::full,
                            "  [{}, {}, {}.{}]\n", device, prop.name, prop.major, prop.minor);
        device_names.emplace_back(prop.name);
    }
    PLSSVM_DETAIL_PERFORMANCE_TRACKER_ADD_TRACKING_ENTRY((plssvm::detail::tracking_entry{ "backend", "device", device_names }));
    plssvm::detail::log(verbosity_level::full | verbosity_level::timing,
                        "\n");
}

void csvm::device_synchronize(const queue_type &queue) const {
    detail::device_synchronize(queue);
}

unsigned long long csvm::get_device_memory() const {
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, devices_[0]);
    return static_cast<unsigned long long>(prop.totalGlobalMem);
}

unsigned long long csvm::get_max_mem_alloc_size() const {
    return this->get_device_memory();
}

std::size_t csvm::get_max_work_group_size() const {
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, devices_[0]);
    return static_cast<std::size_t>(prop.maxThreadsPerBlock);
}

//***************************************************//
//                        fit                        //
//***************************************************//

auto csvm::run_assemble_kernel_matrix_explicit(const parameter &params, const device_ptr_type &data_d, const device_ptr_type &q_red_d, real_type QA_cost) const -> device_ptr_type {
    const unsigned long long num_rows_reduced = data_d.size(0) - 1;
    const unsigned long long num_features = data_d.size(1);

    // define grid and block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    if (max_work_group_size < THREAD_BLOCK_SIZE * THREAD_BLOCK_SIZE) {
        throw kernel_launch_resources{ fmt::format("Not enough work-items allowed for a work-groups of size {}x{}! Try reducing THREAD_BLOCK_SIZE.", THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE) };
    }
    const dim3 block(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_rows_reduced) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                    static_cast<int>(std::ceil(static_cast<double>(num_rows_reduced) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

    device_ptr_type kernel_matrix_d{ num_rows_reduced * (num_rows_reduced + 1) / 2 };  // only explicitly store the upper triangular matrix
    const real_type cost_factor = real_type{ 1.0 } / params.cost;

    detail::set_device(0);
    switch (params.kernel_type) {
        case kernel_function_type::linear:
            cuda::device_kernel_assembly_linear<<<grid, block>>>(kernel_matrix_d.get(), data_d.get(), num_rows_reduced, num_features, q_red_d.get(), QA_cost, cost_factor);
            break;
        case kernel_function_type::polynomial:
            cuda::device_kernel_assembly_polynomial<<<grid, block>>>(kernel_matrix_d.get(), data_d.get(), num_rows_reduced, num_features, q_red_d.get(), QA_cost, cost_factor, params.degree.value(), params.gamma.value(), params.coef0.value());
            break;
        case kernel_function_type::rbf:
            cuda::device_kernel_assembly_rbf<<<grid, block>>>(kernel_matrix_d.get(), data_d.get(), num_rows_reduced, num_features, q_red_d.get(), QA_cost, cost_factor, params.gamma.value());
            break;
    }
    detail::peek_at_last_error();
    this->device_synchronize(devices_[0]);

    return kernel_matrix_d;
}

void csvm::run_gemm_kernel_explicit(const std::size_t m, const std::size_t n, const std::size_t k, const real_type alpha, const device_ptr_type &A_d, const device_ptr_type &B_d, const real_type beta, device_ptr_type &C_d) const {
    // define the grid and block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    if (max_work_group_size < THREAD_BLOCK_SIZE * THREAD_BLOCK_SIZE) {
        throw kernel_launch_resources{ fmt::format("Not enough work-items allowed for a work-groups of size {}x{}! Try reducing THREAD_BLOCK_SIZE.", THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE) };
    }
    const dim3 block(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(n) / static_cast<double>(block.x * INTERNAL_BLOCK_SIZE))),
                    static_cast<int>(std::ceil(static_cast<double>(m) / static_cast<double>(block.y * INTERNAL_BLOCK_SIZE))));

    // cast to correct type
    const auto m_ull = static_cast<unsigned long long>(m);
    const auto n_ull = static_cast<unsigned long long>(n);
    const auto k_ull = static_cast<unsigned long long>(k);

    detail::set_device(0);
    cuda::device_kernel_gemm<<<grid, block>>>(m_ull, n_ull, k_ull, alpha, A_d.get(), B_d.get(), beta, C_d.get());
    detail::peek_at_last_error();
    this->device_synchronize(devices_[0]);
}

//***************************************************//
//                   predict, score                  //
//***************************************************//

auto csvm::run_w_kernel(const device_ptr_type &alpha_d, const device_ptr_type &sv_d) const -> device_ptr_type {
    const unsigned long long num_classes = alpha_d.size(0);
    const unsigned long long num_sv = sv_d.size(0);
    const unsigned long long num_features = sv_d.size(1);

    // define the grid and block sizes
    const std::size_t max_work_group_size = this->get_max_work_group_size();
    const auto max_work_group_size_2D = static_cast<int>(max_work_group_size / 4);
    const dim3 block(max_work_group_size_2D, 4);
    const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_features) / static_cast<double>(block.x))),
                    static_cast<int>(std::ceil(static_cast<double>(num_classes) / static_cast<double>(block.y))));

    device_ptr_type w_d{ { num_classes, num_features } };

    detail::set_device(0);
    cuda::device_kernel_w_linear<<<grid, block>>>(w_d.get(), alpha_d.get(), sv_d.get(), num_classes, num_sv, num_features);
    detail::peek_at_last_error();
    this->device_synchronize(devices_[0]);

    return w_d;
}

auto csvm::run_predict_kernel(const parameter &params, const device_ptr_type &w_d, const device_ptr_type &alpha_d, const device_ptr_type &rho_d, const device_ptr_type &sv_d, const device_ptr_type &predict_points_d) const -> device_ptr_type {
    const unsigned long long num_classes = alpha_d.size(0);
    const unsigned long long num_sv = sv_d.size(0);
    const unsigned long long num_predict_points = predict_points_d.size(0);
    const unsigned long long num_features = predict_points_d.size(1);

    device_ptr_type out_d{ { num_predict_points, num_classes } };

    detail::set_device(0);
    if (params.kernel_type == kernel_function_type::linear) {
        // define the grid and block sizes
        const std::size_t max_work_group_size = this->get_max_work_group_size();
        const auto max_work_group_size_2D = static_cast<int>(max_work_group_size / 4);
        const dim3 block(max_work_group_size_2D, 4);
        const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_predict_points) / static_cast<double>(block.x))),
                        static_cast<int>(std::ceil(static_cast<double>(num_classes) / static_cast<double>(block.y))));

        cuda::device_kernel_predict_linear<<<grid, block>>>(out_d.get(), w_d.get(), rho_d.get(), predict_points_d.get(), num_classes, num_predict_points, num_features);
    } else {
        // define the grid and block sizes
        const std::size_t max_work_group_size = this->get_max_work_group_size();
        const auto max_work_group_size_3D = static_cast<int>(std::sqrt(static_cast<real_type>(max_work_group_size / 4)));
        const dim3 block(max_work_group_size_3D, max_work_group_size_3D, 4);
        const dim3 grid(static_cast<int>(std::ceil(static_cast<double>(num_sv) / static_cast<double>(block.x))),
                        static_cast<int>(std::ceil(static_cast<double>(num_predict_points) / static_cast<double>(block.y))),
                        static_cast<int>(std::ceil(static_cast<double>(num_classes) / static_cast<double>(block.z))));

        switch (params.kernel_type) {
            case kernel_function_type::linear:
                // already handled
                break;
            case kernel_function_type::polynomial:
                cuda::device_kernel_predict_polynomial<<<grid, block>>>(out_d.get(), alpha_d.get(), rho_d.get(), sv_d.get(), predict_points_d.get(), num_classes, num_sv, num_predict_points, num_features, params.degree.value(), params.gamma.value(), params.coef0.value());
                break;
            case kernel_function_type::rbf:
                cuda::device_kernel_predict_rbf<<<grid, block>>>(out_d.get(), alpha_d.get(), rho_d.get(), sv_d.get(), predict_points_d.get(), num_classes, num_sv, num_predict_points, num_features, params.gamma.value());
                break;
        }
    }
    detail::peek_at_last_error();
    this->device_synchronize(devices_[0]);

    return out_d;
}

}  // namespace plssvm::cuda
