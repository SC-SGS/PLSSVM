#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/q_kernel.cuh"

#include "plssvm/constants.hpp"  // plssvm::THREAD_BLOCK_SIZE, plssvm::INTERNAL_BLOCK_SIZE

namespace plssvm::cuda {

template <typename real_type>
__global__ void device_kernel_q_linear(real_type *q, const real_type *data_d, const real_type *data_last, const kernel_index_type num_rows, const kernel_index_type feature_range) {
    const kernel_index_type index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_rows) {
        real_type temp{ 0.0 };
        for (kernel_index_type i = 0; i < feature_range; ++i) {
            temp += data_d[index * feature_range + i] * data_last[i];
        }
        q[index] = temp;
    }
}
template __global__ void device_kernel_q_linear(float *, const float *, const float *, const kernel_index_type, const kernel_index_type);
template __global__ void device_kernel_q_linear(double *, const double *, const double *, const kernel_index_type, const kernel_index_type);

template <typename real_type>
__global__ void device_kernel_q_polynomial(real_type *q, const real_type *data_d, const real_type *data_last, const kernel_index_type num_rows, const kernel_index_type num_cols, const int degree, const real_type gamma, const real_type coef0) {
    const kernel_index_type index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_rows) {
        real_type temp{ 0.0 };
        for (kernel_index_type i = 0; i < num_cols; ++i) {
            temp += data_d[index * num_cols + i] * data_last[i];
        }
        q[index] = pow(gamma * temp + coef0, degree);
    }
}
template __global__ void device_kernel_q_polynomial(float *, const float *, const float *, const kernel_index_type, const kernel_index_type, const int, const float, const float);
template __global__ void device_kernel_q_polynomial(double *, const double *, const double *, const kernel_index_type, const kernel_index_type, const int, const double, const double);

template <typename real_type>
__global__ void device_kernel_q_rbf(real_type *q, const real_type *data_d, const real_type *data_last, const kernel_index_type num_rows, const kernel_index_type num_cols, const real_type gamma) {
    const kernel_index_type index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_rows) {
        real_type temp{ 0.0 };
        for (kernel_index_type i = 0; i < num_cols; ++i) {
            temp += (data_d[index * num_cols + i] - data_last[i]) * (data_d[index * num_cols + i] - data_last[i]);
        }
        q[index] = exp(-gamma * temp);
    }
}
template __global__ void device_kernel_q_rbf(float *, const float *, const float *, const kernel_index_type, const kernel_index_type, const float);
template __global__ void device_kernel_q_rbf(double *, const double *, const double *, const kernel_index_type, const kernel_index_type, const double);

}  // namespace plssvm::cuda
