#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/cg_explicit/kernel_matrix_assembly.cuh"

#include "plssvm/constants.hpp"  // plssvm::real_type, plssvm::THREAD_BLOCK_SIZE, plssvm::FEATURE_BLOCK_SIZE

namespace plssvm::cuda {

__global__ void device_kernel_assembly_linear(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost) {
    const unsigned long long i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;
    const unsigned long long i_linear = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE + threadIdx.x;
    const unsigned long long j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;
    const unsigned long long j_cached_idx_linear = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE + threadIdx.x;

    __shared__ real_type data_cache_i[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];
    __shared__ real_type data_cache_j[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];

    if (blockIdx.x >= blockIdx.y) {
        real_type temp[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };

        for (unsigned long long dim = 0; dim < num_features; dim += FEATURE_BLOCK_SIZE) {
            // zero out shared memory
            for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
                data_cache_i[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
                data_cache_i[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
                data_cache_j[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
                data_cache_j[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            }

            // load data into shared memory
            for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
                const unsigned long long global_i = i_linear + internal * THREAD_BLOCK_SIZE;
                const unsigned long long global_j = j_cached_idx_linear + internal * THREAD_BLOCK_SIZE;

                if (global_i < num_rows) {
                    if (dim + threadIdx.y < num_features) {
                        data_cache_i[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + global_i];
                    }
                    if (dim + threadIdx.y + THREAD_BLOCK_SIZE < num_features) {
                        data_cache_i[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * (num_rows + 1) + global_i];
                    }
                }
                if (global_j < num_rows) {
                    if (dim + threadIdx.y < num_features) {
                        data_cache_j[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + global_j];
                    }
                    if (dim + threadIdx.y + THREAD_BLOCK_SIZE < num_features) {
                        data_cache_j[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * (num_rows + 1) + global_j];
                    }
                }
            }
            __syncthreads();

            // calculation
            for (unsigned block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
                for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
                    for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                        temp[internal_i][internal_j] += data_cache_i[block_dim][threadIdx.x * INTERNAL_BLOCK_SIZE + internal_i] * data_cache_j[block_dim][threadIdx.y * INTERNAL_BLOCK_SIZE + internal_j];
                    }
                }
            }
            __syncthreads();
        }

        for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
            for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                const unsigned long long global_i = i + internal_i;
                const unsigned long long global_j = j + internal_j;

                if (global_i < num_rows && global_j < num_rows && global_i >= global_j) {
                    real_type temp_ij = temp[internal_i][internal_j];
                    temp_ij = temp_ij + QA_cost - q[global_i] - q[global_j];
                    if (global_i == global_j) {
                        temp_ij += cost;
                    }

                    ret[global_j * num_rows + global_i - global_j * (global_j + 1) / 2] = temp_ij;
                }
            }
        }
    }
}

__global__ void device_kernel_assembly_polynomial(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost, const int degree, const real_type gamma, const real_type coef0) {
    const unsigned long long i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;
    const unsigned long long i_linear = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE + threadIdx.x;
    const unsigned long long j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;
    const unsigned long long j_cached_idx_linear = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE + threadIdx.x;

    __shared__ real_type data_cache_i[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];
    __shared__ real_type data_cache_j[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];

    if (blockIdx.x >= blockIdx.y) {
        real_type temp[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };

        for (unsigned long long dim = 0; dim < num_features; dim += FEATURE_BLOCK_SIZE) {
            // zero out shared memory
            for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
                data_cache_i[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
                data_cache_i[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
                data_cache_j[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
                data_cache_j[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = real_type{ 0.0 };
            }

            // load data into shared memory
            for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
                const unsigned long long global_i = i_linear + internal * THREAD_BLOCK_SIZE;
                const unsigned long long global_j = j_cached_idx_linear + internal * THREAD_BLOCK_SIZE;

                if (global_i < num_rows) {
                    if (dim + threadIdx.y < num_features) {
                        data_cache_i[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + global_i];
                    }
                    if (dim + threadIdx.y + THREAD_BLOCK_SIZE < num_features) {
                        data_cache_i[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * (num_rows + 1) + global_i];
                    }
                }
                if (global_j < num_rows) {
                    if (dim + threadIdx.y < num_features) {
                        data_cache_j[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1) + global_j];
                    }
                    if (dim + threadIdx.y + THREAD_BLOCK_SIZE < num_features) {
                        data_cache_j[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * (num_rows + 1) + global_j];
                    }
                }
            }
            __syncthreads();

            // calculation
            for (unsigned block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
                for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
                    for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                        temp[internal_i][internal_j] += data_cache_i[block_dim][threadIdx.x * INTERNAL_BLOCK_SIZE + internal_i] * data_cache_j[block_dim][threadIdx.y * INTERNAL_BLOCK_SIZE + internal_j];
                    }
                }
            }
            __syncthreads();
        }

        for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
            for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                const unsigned long long global_i = i + internal_i;
                const unsigned long long global_j = j + internal_j;

                if (global_i < num_rows && global_j < num_rows && global_i >= global_j) {
                    real_type temp_ij = temp[internal_i][internal_j];
                    temp_ij = pow(gamma * temp_ij + coef0, (double) degree) + QA_cost - q[global_i] - q[global_j];
                    if (global_i == global_j) {
                        temp_ij += cost;
                    }

                    ret[global_j * num_rows + global_i - global_j * (global_j + 1) / 2] = temp_ij;
                }
            }
        }
    }
}

__global__ void device_kernel_assembly_rbf(real_type *ret, const real_type *data_d, const unsigned long long num_rows, const unsigned long long num_features, const real_type *q, const real_type QA_cost, const real_type cost, const real_type gamma) {
    const unsigned long long i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;
    const unsigned long long i_linear = blockIdx.x * blockDim.x * INTERNAL_BLOCK_SIZE + threadIdx.x;
    const unsigned long long j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;
    const unsigned long long j_cached_idx_linear = blockIdx.y * blockDim.y * INTERNAL_BLOCK_SIZE + threadIdx.x;

    __shared__ real_type data_cache_i[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];
    __shared__ real_type data_cache_j[FEATURE_BLOCK_SIZE][INTERNAL_BLOCK_SIZE * THREAD_BLOCK_SIZE];

    if (blockIdx.x >= blockIdx.y) {
        constexpr unsigned PADDING = THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE;
        real_type temp[INTERNAL_BLOCK_SIZE][INTERNAL_BLOCK_SIZE] = { 0.0 };

        for (unsigned long long dim = 0; dim < num_features; dim += FEATURE_BLOCK_SIZE) {
            // load data into shared memory
            for (unsigned internal = 0; internal < INTERNAL_BLOCK_SIZE; ++internal) {
                const unsigned long long global_i = i_linear + internal * THREAD_BLOCK_SIZE;
                const unsigned long long global_j = j_cached_idx_linear + internal * THREAD_BLOCK_SIZE;

                data_cache_i[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1 + PADDING) + global_i];
                data_cache_i[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * (num_rows + 1 + PADDING) + global_i];
                data_cache_j[threadIdx.y][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y) * (num_rows + 1 + PADDING) + global_j];
                data_cache_j[threadIdx.y + THREAD_BLOCK_SIZE][internal * THREAD_BLOCK_SIZE + threadIdx.x] = data_d[(dim + threadIdx.y + THREAD_BLOCK_SIZE) * (num_rows + 1 + PADDING) + global_j];
            }
            __syncthreads();

            // calculation
            for (unsigned block_dim = 0; block_dim < FEATURE_BLOCK_SIZE; ++block_dim) {
                for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
                    for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                        const real_type d = data_cache_i[block_dim][threadIdx.x * INTERNAL_BLOCK_SIZE + internal_i] - data_cache_j[block_dim][threadIdx.y * INTERNAL_BLOCK_SIZE + internal_j];
                        temp[internal_i][internal_j] += d * d;
                    }
                }
            }
            __syncthreads();
        }

        for (unsigned internal_i = 0; internal_i < INTERNAL_BLOCK_SIZE; ++internal_i) {
            for (unsigned internal_j = 0; internal_j < INTERNAL_BLOCK_SIZE; ++internal_j) {
                const unsigned long long global_i = i + internal_i;
                const unsigned long long global_j = j + internal_j;

                // TODO: remove global_i < num_rows && global_j < num_rows? -> padding entries may not all be zero in ret afterwards -> adjust BLAS kernel
                if (global_i < num_rows && global_j < num_rows && global_i >= global_j) {
                    real_type temp_ij = temp[internal_i][internal_j];
                    temp_ij = exp(-gamma * temp_ij) + QA_cost - q[global_i] - q[global_j];
//                    temp_ij += (global_i == global_j) * cost;
                    if (global_i == global_j) {
                        temp_ij += cost;
                    }

                    ret[global_j * (num_rows + PADDING) + global_i - global_j * (global_j + 1) / 2] = temp_ij;
                }
            }
        }
    }
}

}  // namespace plssvm::cuda